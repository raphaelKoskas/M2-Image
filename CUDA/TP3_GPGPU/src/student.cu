#include "hip/hip_runtime.h"
/*
* TP 3 - Réduction CUDA
* --------------------------
* Mémoire paratagée, synchronisation, optimisation
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"

namespace IMAC
{
	
	// ==================================================== EX 1
    __global__
    void maxReduce_ex1(const uint *const dev_array, const uint size, uint *const dev_partialMax)
	{
		extern __shared__ uint sharedMemory[];
		uint idx = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int i=1;

		sharedMemory[threadIdx.x] = idx < size ? dev_array[idx] : 0;
		__syncthreads();

		while (2 * i * threadIdx.x +i  < blockDim.x){
			sharedMemory[2 * i * threadIdx.x ] = umax(sharedMemory[2 * i * threadIdx.x],sharedMemory[2 * i * threadIdx.x +i]);
			__syncthreads();
			i*=2;
		}
		if (threadIdx.x==0){
			dev_partialMax[blockIdx.x]=sharedMemory[0];
		}
	}
	__global__
    void maxReduce_ex2(const uint *const dev_array, const uint size, uint *const dev_partialMax)
	{
		extern __shared__ uint sharedMemory[];
		uint idx = blockDim.x * blockIdx.x + threadIdx.x;
		uint i= blockDim.x >> 1;
		bool keepGoing = threadIdx.x+1  <= i;
		sharedMemory[threadIdx.x] = idx < size ? dev_array[idx] : 0;
		__syncthreads();
		while (keepGoing){
			sharedMemory[threadIdx.x ] = umax(sharedMemory[threadIdx.x],sharedMemory[threadIdx.x + i]);
			i >>= 1;
			keepGoing = threadIdx.x+1  <= i;
			__syncthreads();
		}
		if (threadIdx.x==0){
			dev_partialMax[blockIdx.x]=sharedMemory[0];
		}
	}

	__global__
    void maxReduce_ex3(const uint *const dev_array, const uint size, uint *const dev_partialMax)
	{
		extern __shared__ uint sharedMemory[];
		uint idx = blockDim.x * (2*blockIdx.x) + threadIdx.x;
		uint i= blockDim.x >> 1;
		bool keepGoing = threadIdx.x+1  <= i;
		sharedMemory[threadIdx.x] = idx < size ? umax(dev_array[idx],dev_array[idx+blockDim.x]) : 0;
		__syncthreads();
		while (keepGoing){
			sharedMemory[threadIdx.x ] = umax(sharedMemory[threadIdx.x],sharedMemory[threadIdx.x + i]);
			i >>= 1;
			keepGoing = threadIdx.x+1  <= i;
			__syncthreads();
		}
		if (threadIdx.x==0){
			dev_partialMax[blockIdx.x]=sharedMemory[0];
		}
	}

	__global__
    void maxReduce_ex4(const uint *const dev_array, const uint size, uint *const dev_partialMax)
	{
		extern __shared__ uint sharedMemory[];
		uint idx = blockDim.x * (2*blockIdx.x) + threadIdx.x;
		uint i= blockDim.x >> 1;
		bool keepGoing = threadIdx.x+1  <= i;
		sharedMemory[threadIdx.x] = idx < size ? umax(dev_array[idx],dev_array[idx+blockDim.x]) : 0;
		__syncthreads();
		while (keepGoing){
			sharedMemory[threadIdx.x ] = umax(sharedMemory[threadIdx.x],sharedMemory[threadIdx.x + i]);
			i >>= 1;
			if (i<=32) break;
			keepGoing = threadIdx.x+1  <= i;
			__syncthreads();
		}
		if (threadIdx.x < 32){
			volatile uint *sharedMem = sharedMemory;
			if (blockDim.x >= 64) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 32]);
			if (blockDim.x >= 32) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 16]);
			if (blockDim.x >= 16) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 8]);
			if (blockDim.x >= 8) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 4]);
			if (blockDim.x >= 4) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 2]);
			if (blockDim.x >= 2) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 1]);
		}
		if (threadIdx.x==0){
			dev_partialMax[blockIdx.x]=sharedMemory[0];
		}
	}

	template <unsigned int N>
	__global__
    void maxReduce_ex5(const uint *const dev_array, const uint size, uint *const dev_partialMax)
	{
		extern __shared__ uint sharedMemory[];
		uint idx = blockDim.x * (2*blockIdx.x) + threadIdx.x;
		sharedMemory[threadIdx.x] = idx < size ? umax(dev_array[idx],dev_array[idx+blockDim.x]) : 0;
		__syncthreads();
		volatile uint *sharedMem = sharedMemory;
		if (blockDim.x >= 1024 && threadIdx.x < 512) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 512]);
		if (blockDim.x >= 512 && threadIdx.x < 256) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 256]);
		if (blockDim.x >= 256 && threadIdx.x < 128) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 128]);
		if (blockDim.x >= 128 && threadIdx.x < 64) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 64]);
		if (blockDim.x >= 64 && threadIdx.x < 32) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 32]);
		if (blockDim.x >= 32 && threadIdx.x < 16) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 16]);
		if (blockDim.x >= 16 && threadIdx.x < 8) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 8]);
		if (blockDim.x >= 8 && threadIdx.x < 4) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 4]);
		if (blockDim.x >= 4 && threadIdx.x < 2) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 2]);
		if (blockDim.x >= 2 && threadIdx.x < 1) sharedMem[threadIdx.x] = umax(sharedMem[threadIdx.x],sharedMem[threadIdx.x + 1]);
		if (threadIdx.x==0){
			dev_partialMax[blockIdx.x]=sharedMemory[0];
		}
	}

	void studentJob(const std::vector<uint> &array, const uint resCPU /* Just for comparison */, const uint nbIterations)
    {
		uint *dev_array = NULL;
        const size_t bytes = array.size() * sizeof(uint);

		// Allocate array on GPU
		HANDLE_ERROR( hipMalloc( (void**)&dev_array, bytes ) );
		// Copy data from host to device
		HANDLE_ERROR( hipMemcpy( dev_array, array.data(), bytes, hipMemcpyHostToDevice ) );

		std::cout << "Test with " << nbIterations << " iterations" << std::endl;

		std::cout << "========== Ex 1 " << std::endl;
		uint res1 = 0; // result
		// Launch reduction and get timing
		float2 timing1 = reduce<KERNEL_EX1>(nbIterations, dev_array, array.size(), res1);
		
        std::cout << " -> Done: ";
        printTiming(timing1);
		compare(res1, resCPU); // Compare results

		std::cout << "========== Ex 2 " << std::endl;
		uint res2 = 0; // result
		// Launch reduction and get timing
		float2 timing2 = reduce<KERNEL_EX2>(nbIterations, dev_array, array.size(), res2);
		
        std::cout << " -> Done: ";
        printTiming(timing2);
		compare(res2, resCPU);

		std::cout << "========== Ex 3 " << std::endl;
		uint res3 = 0; // result
		// Launch reduction and get timing
		float2 timing3 = reduce<KERNEL_EX3>(nbIterations, dev_array, array.size(), res3);
		
        std::cout << " -> Done: ";
        printTiming(timing3);
		compare(res3, resCPU);

		std::cout << "========== Ex 4 " << std::endl;
		uint res4 = 0; // result
		// Launch reduction and get timing
		float2 timing4 = reduce<KERNEL_EX4>(nbIterations, dev_array, array.size(), res4);
		
        std::cout << " -> Done: ";
        printTiming(timing4);
		compare(res4, resCPU);

		std::cout << "========== Ex 5 " << std::endl;
		uint res5 = 0; // result
		// Launch reduction and get timing
		float2 timing5 = reduce<KERNEL_EX5>(nbIterations, dev_array, array.size(), res5);
		
        std::cout << " -> Done: ";
        printTiming(timing5);
		compare(res5, resCPU);

		// Free array on GPU
		hipFree( dev_array );
    }

	void printTiming(const float2 timing)
	{
		std::cout << ( timing.x < 1.f ? 1e3f * timing.x : timing.x ) << " us on device and ";
		std::cout << ( timing.y < 1.f ? 1e3f * timing.y : timing.y ) << " us on host." << std::endl;
	}

    void compare(const uint resGPU, const uint resCPU)
	{
		if (resGPU == resCPU)
		{
			std::cout << "Well done ! " << resGPU << " == " << resCPU << " !!!" << std::endl;
		}
		else
		{
			std::cout << "You failed ! " << resGPU << " != " << resCPU << " !!!" << std::endl;
		}
	}
}